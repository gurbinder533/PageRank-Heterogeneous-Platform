#include "hip/hip_runtime.h"
/***
 * Author: Gurbinder Gill
 * Email : gill@cs.utexas.edu
 */
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <mpi.h>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "../../include/custom_util_cuda.h"

using namespace std;

__global__ void add1kernel(int* input)
{
  int tid = blockIdx.x*blockDim.x +  threadIdx.x;
  input[tid] = input[tid] + 1;
}

__global__ void checkNodes_kernel(Node* local_nodes, unsigned num_nodes)
{
  int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < num_nodes)
  {
    local_nodes[index].rank = 1;
    printf("on device : %d\n", local_nodes[index].rank);
  }
}


__global__ void PageRank_init(Node* local_nodes,unsigned* out_edges, unsigned num_nodes, float* delta_send, int host_id)
{
  int index = threadIdx.x + blockDim.x*blockIdx.x;

  if(index < num_nodes)
  {
    Node n = local_nodes[index];
    float old_residual = n.rank*alpha;

    if(n.numOutEdges != 0)
    {
      float delta = old_residual/n.numOutEdges;
      for(int ii = n.s_index; ii < (n.s_index + n.numOutEdges); ++ii)
      {
        atomicAdd(&delta_send[out_edges[ii]], delta);
      }
    }
  }

}


__global__ void PageRank(Node* local_nodes,unsigned* out_edges, unsigned num_nodes, float* delta_send, int host_id)
{
  int index = threadIdx.x + blockDim.x*blockIdx.x;

  if(index < num_nodes)
  {
    Node n = local_nodes[index];
    float old_residual = n.residual;
    n.residual = 0.0;

    local_nodes[index].rank += old_residual;

    if(n.numOutEdges !=0)
    {
      float delta = old_residual*alpha/n.numOutEdges;
      for(int ii = n.s_index; ii < (n.s_index + n.numOutEdges); ++ii)
      {
        atomicAdd(&delta_send[out_edges[ii]], delta);
      }
    }
  }

}

__global__ void ApplyDelta(Node* local_nodes, float* delta_recv, float* delta_send, unsigned total_nodes, unsigned global_chunk_size, unsigned hosts, unsigned my_id)
{
  int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < global_chunk_size)
  {
    int j = 0;
    float temp_residual = 0.0;
    for(int i = 0; i < hosts; ++i)
    {
        j = index + global_chunk_size*i;
        temp_residual += delta_recv[j];
        delta_send[j] = 0.0;
    }
    local_nodes[index].residual = temp_residual;
  }
}

__global__ void finish_pageRank(Node* local_nodes, unsigned total_nodes, unsigned local_chunk_size)
{

  int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < local_chunk_size)
  {
    local_nodes[index].rank = local_nodes[index].rank/total_nodes;
  }
}

void cudaFunction(unsigned total_nodes,unsigned global_chunk_size, vector<Node>& local_nodes, vector<unsigned>& out_edges, int iterations)
{

  int local_chunk_size = local_nodes.size();

  std::cout << "local chunk size :" << local_chunk_size << "\n";
  std::cout << "Global chunk size :" << global_chunk_size << "\n";

  int my_id;
  int hosts;
  MPI_Comm_rank(MPI_COMM_WORLD, &my_id);
  MPI_Comm_size(MPI_COMM_WORLD, &hosts);

  std::cout << " ID : " << my_id << "\n";
  Node* device_local_nodes;

  hipMalloc(&device_local_nodes, sizeof(Node)*local_chunk_size);
  hipMemcpy(device_local_nodes, &local_nodes.front(), sizeof(Node)*local_chunk_size, hipMemcpyHostToDevice);

  int num_outEdges = out_edges.size();
  unsigned* device_out_edges;

  hipMalloc(&device_out_edges, sizeof(unsigned)*num_outEdges);
  hipMemcpy(device_out_edges, &out_edges.front(), sizeof(unsigned)*num_outEdges, hipMemcpyHostToDevice);


  float *device_delta_send, *device_delta_recv;
  unsigned total_new = hosts*global_chunk_size;
  hipMalloc(&device_delta_send, sizeof(float)*total_new);
  hipMalloc(&device_delta_recv, sizeof(float)*total_new);
  float *host_zero;
  host_zero = (float*)calloc(total_new, sizeof(float));

  hipMemcpy(device_delta_send, host_zero, sizeof(float)*total_new, hipMemcpyHostToDevice);
  hipMemcpy(device_delta_recv, host_zero, sizeof(float)*total_new, hipMemcpyHostToDevice);



  // Bulk synchronous PageRank : STARTS

  int threads = 512;
  int blocks = local_chunk_size/threads + (local_chunk_size%threads == 0 ? 0 : 1);

  cout << "Blocks : " << blocks << "\n";

  ///////////////// Intialize Graph ///////////////////////

  if(my_id == 0)
    std::cout << " PAGE RANK : Initialization Phase Starts\n";
  PageRank_init<<<blocks, threads>>>(device_local_nodes, device_out_edges, local_chunk_size, device_delta_send,my_id );
  hipDeviceSynchronize();

  // Using MPI_AllToAll
  int status;
  status = MPI_Alltoall(device_delta_send,global_chunk_size, MPI_FLOAT, device_delta_recv, global_chunk_size, MPI_FLOAT, MPI_COMM_WORLD);

  if(status != MPI_SUCCESS)
  {
    cout << "MPI_ALLToAll failed\n";
    exit(0);
  }


  // Apply received delta values.
  ApplyDelta<<<blocks, threads>>>(device_local_nodes, device_delta_recv, device_delta_send, total_nodes, global_chunk_size, hosts, my_id );
  hipDeviceSynchronize();


  if(my_id == 0)
    std::cout << " PAGE RANK : Initialization Phase Ends\n";
  //////////////////////////////////////////////////////


  int iterations_fixed = 60;
  clock_t start_pg, end_pg;
  start_pg = clock();

  // Different timers
  //clock_t pageRank_start_t, mpi_start_t, apply_start_t;
  //double pageRank_dur_t = 0, mpi_dur_t = 0, apply_dur_t = 0;

  for(int itr = 0; itr < iterations_fixed; ++itr)
  {
    if(my_id == 0)
      std::cout << "Iteration no. : " << itr <<"\n";

    //pageRank_start_t  = clock();

    PageRank<<<blocks, threads>>>(device_local_nodes, device_out_edges, local_chunk_size, device_delta_send,my_id );
    hipDeviceSynchronize();

    //pageRank_dur_t +=  double(clock() - pageRank_start_t) / CLOCKS_PER_SEC;

    // CUDA Aware MPI
    int status;

    //mpi_start_t = clock();

    // Using MPI_AllToAll
    status = MPI_Alltoall(device_delta_send,global_chunk_size, MPI_FLOAT, device_delta_recv, global_chunk_size, MPI_FLOAT, MPI_COMM_WORLD);

    if(status != MPI_SUCCESS)
    {
      cout << "MPI_ALLToAll failed\n";
      exit(0);
    }

    //mpi_dur_t +=  double(clock() - mpi_start_t) / CLOCKS_PER_SEC;


    //apply_start_t = clock();

    // Apply received delta values.
    ApplyDelta<<<blocks, threads>>>(device_local_nodes, device_delta_recv, device_delta_send, total_nodes, global_chunk_size, hosts, my_id );
    hipDeviceSynchronize();

    //apply_dur_t +=  double(clock() - apply_start_t) / CLOCKS_PER_SEC;

    // PageRank : ENDS
  }

  end_pg = clock();
  double elapsed_secs = double(end_pg - start_pg) / CLOCKS_PER_SEC;

  if(my_id == 0)
  {
    std::cout << "Total Time for " << iterations << " : " << elapsed_secs << " sec.\n";
   // std::cout << "Total Time for  PageRank routine: " << pageRank_dur_t << " sec.\n";
   // std::cout << "Total Time for  MPI routine: " << mpi_dur_t << " sec.\n";
   // std::cout << "Total Time for  Apply routine: " << apply_dur_t << " sec.\n";

    Node * host_check_nodes;
    host_check_nodes = (Node*)malloc(sizeof(Node)*local_chunk_size);
    hipMemcpy(host_check_nodes, device_local_nodes, sizeof(Node)*local_chunk_size, hipMemcpyDeviceToHost);

    for(int i = 0; i < 10; ++i)
    {
        cout << "\n R : " << i << " : " << host_check_nodes[i].rank << "\n";
    }
  }


}

